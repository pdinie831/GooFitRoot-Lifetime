#include "hip/hip_runtime.h"
//
//
//
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <sys/stat.h>
#include "Riostream.h"
#include <map>
#include <string>
#include <vector>
#include <math.h>
#include <TCint.h>
#include <TGenericClassInfo.h> 
#include <TH1.h>
#include <TH2.h>
#include <TH1F.h>
#include <TH2F.h>
#include <TH1D.h>
#include <TH2D.h>
#include <TROOT.h>
#include <TEnv.h>
#include <TSystem.h>
#include <TTree.h>
#include "TBranch.h"
#include <TApplication.h>
#include <TFile.h>
#include <TCanvas.h>
#include <TStyle.h> 
#include <TF1.h>  
#include <TF2.h> 
#include <TLorentzVector.h>
#include <TVector3.h>
#include "TDSet.h"
#include "TChain.h"
#include <time.h> 
#include <TSystemDirectory.h>
#include <TLegend.h>
#include "FitManager.hh"
#include "UnbinnedDataSet.hh" 
#include "LandauPdf.hh" 
#include "NovosibirskPdf.hh"
#include "BifurGaussPdf.hh" 
#include "SimpleCheby2Pdf.hh" 

#include "TRandom.h" 
#if HAVE_ROOT
#  include "Variable.hh"
#  include "TH1F.h"
//#  include "TH2F.h"			// unused?
#  include "TStyle.h"
#  include "TCanvas.h"
#else
#  include "fakeTH1F.h"
#endif

#include <sys/time.h>
#include <sys/times.h>
#include <iostream>
// GooFit stuff
#include "Variable.hh" 
#include "KinLimitBWPdf.hh" 
#include "ConvolutionPdf.hh"
#include "GaussianPdf.hh"
#include "ScaledGaussianPdf.hh"
#include "ArgusPdf.hh"
#include "AddPdf.hh"
#include "PolynomialPdf.hh" 
#include "FitManager.hh" 
#include "ExpGausPdf.hh" 
#include "ExpGausPEEPdf.hh" 
#include "ExpPdf.hh" 
#include "ProdPdf.hh" 
#include "RGaussianPdf.hh"
#include "BifurGaussPdf.hh"
#include "ExpGausMPdf.hh" 
#include "ExpGausWithIntPdf.hh"
#include "ExpGausPEEfixSigmaPdf.hh" 
#include "ExpGausProdBPdf.hh"
#include "ExpGausProdEffiBPdf.hh"
#include "ExpGausPEESigmaBPdf.hh" 
#include "PolyEffiPdf.hh" 
#include "ErfcPolyPdf.hh"
#include "SigmoidBpPdf.hh"
#include "SigmoidGausPdf.hh"
// System stuff
#include <fstream> 
#include <sys/time.h>
#include <sys/times.h>

timeval startTime, stopTime, totalTime;
clock_t startCPU, stopCPU; 
tms startProc, stopProc; 

void fitTauSBModel();
using namespace std; 


int main (int argc, char** argv) {

  TApplication app("App",&argc, argv);

  fitTauSBModel(); 
//  app.Run() ;
  cout<<"esco..." <<endl;
   return 0 ;
}


void fitTauSBModel(){


  gettimeofday(&startTime, NULL);
  startCPU = times(&startProc);

   gROOT ->Reset();
   gROOT->SetStyle("Plain");
   gStyle->SetOptStat(000000);
   gStyle->SetOptFit(000000);


   TCanvas* c1 = new TCanvas("c1","Mass PLOTS",200,10,900,780);
   TCanvas* c2 = new TCanvas("c2","cTau PLOTS",200,10,900,780);
   TCanvas* c3 = new TCanvas("c3","STau PLOTS",200,10,900,780);
    Char_t    InputFileName[300] = "test-ntupla.root";
  Char_t    InputTauBpTreeName[10]   = "TauBpTree";
  TFile*InputFile = TFile::Open(InputFileName,"READ","ROOT file");
  
   Char_t    OutFileName[300] =  "testGoofit3DBp-2016.root";
   gSystem->Exec(Form("rm %s",OutFileName));
   TFile*OutFile = TFile::Open(OutFileName,"RECREATE");
   
   float PlotLineWidth = 1.2;
   float MarkerSize    = 0.35;
 
  double xBpMass;
//  double xBpTau;
  double xBpcTau;
  double xSBpcTau;
//  double c_const       = 0.0299792458;
//     double XMinSign = 5.12;
//     double XMaxSign = 5.44;
  double XMinSign = 5.15;
  double XMaxSign = 5.40;
  double BpMass   = 5.279;
//double BpSigma  = 0.020;
  double BpSigma  = 0.022;

  double NSigmaSB = 3.6;
  double BiasSB   = 6;
 
//      double XMinSign = 5.12;
//      double XMaxSign = 5.44;
//double XMinSign = 5.15;
//double XMaxSign = 5.40;
//   double XMinSBL = 4.879000;
//   double XMaxSBL = 5.159000;
//   double XMinSBR = 5.399000;
//   double XMaxSBR = 5.679000;
  double XMinSBL = BpMass -(BiasSB+NSigmaSB)*BpSigma;
  double XMaxSBL = BpMass - BiasSB          *BpSigma;
  double XMinSBR = BpMass + BiasSB          *BpSigma;
  double XMaxSBR = BpMass +(BiasSB+NSigmaSB)*BpSigma;
  double XMin = 0.006;
  double XMax = 0.34;
  double SXMin = 0.0003;
  double SXMax = 0.005;
//  double SXMax = 0.0079;
//  double SXMax = 0.0079;
  double XStepSign = 0.001;
  double XStepcTau = 0.001;
  double XStepScTau = 0.00002;
  double XStepMinuit = 0.00001;
  double XHScale = 4;
  
  double c_const       = 0.0299792458;

  printf("(xBpMass>%8f && xBpMass<%8f || xBpMass>%8f && xBpMass<%8f)",XMinSBL,XMaxSBL,XMinSBR,XMaxSBR);

  Variable* xMass  = new Variable("xMass",XMinSign, XMaxSign); 
  xMass->numbins = (XMaxSign -XMinSign)/XStepSign;
  TH1F HxMass( "HxMass" , "B^{+} Mass"    ,	     xMass->numbins, xMass->lowerlimit, xMass->upperlimit);
  TH1F pdfHist("pdfHist", "B^{+} Mass Fit",  XHScale*xMass->numbins, xMass->lowerlimit, xMass->upperlimit);
  TH1F sigHist("sigHist", "B^{+} Mass Fit",  XHScale*xMass->numbins, xMass->lowerlimit, xMass->upperlimit);
  TH1F bkgHist("bgkHist", "B^{+} Mass Fit",  XHScale*xMass->numbins, xMass->lowerlimit, xMass->upperlimit);

  Variable* xcTau  = new Variable("xcTau",XMin, XMax); 
  xcTau->numbins   = (XMax -XMin)/XStepcTau;
  Variable* xScTau = new Variable("xScTau",SXMin, SXMax); 
  xScTau->numbins = (SXMax -SXMin)/(XStepScTau);
  std::cout<<"xMass ->numbins = "<<xMass ->numbins<<std::endl;
  std::cout<<"xcTau ->numbins = "<<xcTau ->numbins<<std::endl;
  std::cout<<"xScTau->numbins = "<<xScTau->numbins<<std::endl;
  std::cout<<"xScTau->numbins = "<<xScTau->numbins<<std::endl;
 
  TH1F HxcTau(    "HxcTau"   , "B^{+} cTau",          xcTau  ->numbins, xcTau ->lowerlimit,  xcTau ->upperlimit);
  TH1F HxcTauSB(  "HxcTauSB" , "B^{+} cTau SB",       xcTau  ->numbins, xcTau ->lowerlimit,  xcTau ->upperlimit);
  TH1F HxScTau(   "HxScTau"  , "B^{+} cTau Sigma",    xScTau  ->numbins, xScTau->lowerlimit, xScTau ->upperlimit);
  TH1F HxScTauSB( "HxScTauSB", "B^{+} cTau Sigma SB", xScTau  ->numbins, xScTau->lowerlimit, xScTau ->upperlimit);
  TH2F pdf_cTauSTau_Hist2D( "pdf_cTauSTau_Hist2D" , "Bc+ cTau model   pdf",  XHScale*xcTau ->numbins, xcTau ->lowerlimit, xcTau ->upperlimit, XHScale*xScTau->numbins, xScTau ->lowerlimit, xScTau ->upperlimit);
  TH2F sig_cTauSTau_Hist2D( "sig_cTauSTau_Hist2D" , "Bc+ cTau model   sig",  XHScale*xcTau ->numbins, xcTau ->lowerlimit, xcTau ->upperlimit, XHScale*xScTau->numbins, xScTau ->lowerlimit, xScTau ->upperlimit);
  TH2F bkg_cTauSTau_Hist2D( "bkg_cTauSTau_Hist2D" , "Bc+ cTau model   bkg",  XHScale*xcTau ->numbins, xcTau ->lowerlimit, xcTau ->upperlimit, XHScale*xScTau->numbins, xScTau ->lowerlimit, xScTau ->upperlimit);
//
// Mass Spectrum
//
  Variable* mean   = new Variable("mean"  ,5.2785,XStepMinuit, 5., 5.5);
  Variable* sigma1 = new Variable("sigma1",0.0139,XStepMinuit, 0., 1.);
  Variable* sigma2 = new Variable("sigma2",0.0228,XStepMinuit, 0., 1.);
  Variable* sigma3 = new Variable("sigma3",0.0601,XStepMinuit, 0., 1.);

//   GaussianPdf* gauss1 = new GaussianPdf("gauss1", xMass, mean, sigma1);
//   GaussianPdf* gauss2 = new GaussianPdf("gauss2", xMass, mean, sigma2);
//   GaussianPdf* gauss3 = new GaussianPdf("gauss3", xMass, mean, sigma3);
  RGaussianPdf* gauss1 = new RGaussianPdf("gauss1", xMass, mean, sigma1);
  RGaussianPdf* gauss2 = new RGaussianPdf("gauss2", xMass, mean, sigma2);
  RGaussianPdf* gauss3 = new RGaussianPdf("gauss3", xMass, mean, sigma3);

/*   Variable* meanBckgBp   = new Variable("meanBckgBp" ,5.360,0.00001, 5., 5.5);
  Variable* sigmaBckgBp  = new Variable("sigmaBckgBp",0.030,0.00001, 0. , 1. );
  Variable* meanBckgB0   = new Variable("meanBckgB0" ,5.090,0.00001, 5. , 5.2);
  Variable* sigmaBckgB0  = new Variable("sigmaBckgB0",0.025,0.00001, 0. , 1.);
 */  
//   Variable* meanBckgBp   = new Variable("meanBckgBp" ,5.37 ,0,  5.2, 5.7);
//   Variable* sigmaBckgBp  = new Variable("sigmaBckgBp",0.033,0,  0.013 , 0.05 );
//   Variable* meanBckgB0   = new Variable("meanBckgB0" ,5.090,0,  5.0 , 5.15);
//   Variable* sigmaBckgB0  = new Variable("sigmaBckgB0",0.025,0,  0.01 , 0.05);
  Variable* meanBckgBp   = new Variable("meanBckgBp" ,5.355,0, 5.2, 5.7);
  Variable* sigmaBckgBp  = new Variable("sigmaBckgBp",0.020,0, 0.013 , 0.05 );
  Variable* meanBckgB0   = new Variable("meanBckgB0" ,5.010,0, 5.0 , 5.15);
  Variable* sigmaBckgB0  = new Variable("sigmaBckgB0",0.029,0, 0.02 , 0.04);
  Variable* wb1 = new Variable("wb1",1.75628e-01, 0., 1.);
  Variable* wb2 = new Variable("wb2",1.17246e-01, 0., 1.);
  Variable* wb3 = new Variable("wb3",0.1, 0., 1.);

  RGaussianPdf* gaussBckgBp = new RGaussianPdf("gaussBckgBp", xMass, meanBckgBp, sigmaBckgBp);
  RGaussianPdf* gaussBckgB0 = new RGaussianPdf("gaussBckgB0", xMass, meanBckgB0, sigmaBckgB0);
  

  Variable* wg1 = new Variable("wg1",0.44, 0., 1.);
  Variable* wg2 = new Variable("wg2",0.5 , 0., 1.);
  Variable* signalYield = new Variable("signalYield",8.88646e+05,      100000., 2000000.);
  Variable* bckgYield   = new Variable("bckgYield"  ,1.25481e+05 ,      10000., 400000.);

//   Variable* constaCoef = new Variable("constaCoef", 70, 0.001, 10, 100); 
//   Variable* linearCoef = new Variable("linearCoef", 0.1, 0.001, -0.35, 10.); 
//   Variable* secondCoef = new Variable("secondCoef", 0.1, 0.001, 0, 10);
//   Variable* thirdCoef  = new Variable("thirdCoef" , 0.1, 0.001, 0, 10);
 
//   Variable* constaCoef = new Variable("constaCoef", 70., XStepMinuit, 20., 1000); 
//   Variable* linearCoef = new Variable("linearCoef", 0.1, XStepMinuit, -3.5, 10.); 
//   Variable* secondCoef = new Variable("secondCoef", 0.1, XStepMinuit, 0, 10);
//   Variable* thirdCoef  = new Variable("thirdCoef" , 0.1, XStepMinuit, 0, 10);
  Variable* constaCoef = new Variable("constaCoef", 1. ,XStepMinuit,0.,1000. ); 
  Variable* linearCoef = new Variable("linearCoef", 0.001,XStepMinuit,0,10 ); 

  Variable* p0   = new Variable("p0", -6.86109e-01,-10.,10. ); 
  Variable* p1   = new Variable("p1", 0,0,-10,10 ); 
  Variable* VMinSign = new Variable("VMinSign",XMinSign ); 
  Variable* VMaxSign = new Variable("VMaxSign",XMaxSign ); 
  SimpleCheby2Pdf* SimpleCheby2  = new SimpleCheby2Pdf("SimpleCheby2", xMass, p0, p1,VMinSign,VMaxSign);

// double  fullRange = XMaxSign - XMinSign;
// double  minScaled = -1. + 2. * (XMinSign - xminfull) / fullRange;
// 
// double  maxScaled = +1. - 2. * (xmaxfull - XMaxSign)) / fullRange; 

//  Variable* aslope     = new Variable("slope", -1.);
  //Variable* aslope     = new Variable("slope", 0.39, -10, 10);
  //Variable* apower     = new Variable("apower", 6, 0, 10);
//  Variable* apower     = new Variable("apower", 1.18, XStepMinuit, 0.9, 15.);
//  Variable* apower     = new Variable("apower", 1.18, XStepMinuit, 0.9, 6.);
//  Variable* apower     = new Variable("apower", 1.18, 0.001, 0.9, 5.);
  //Variable* treshold   = new Variable("treshold" ,5.168,XStepMinuit, 5.02, 6.);
//  Variable* treshold   = new Variable("treshold" ,5.33,0, 5.04, 6.);

 
 
  std::vector<Variable*> weightsSignalMass;
  weightsSignalMass.push_back(wg1);
//  weightsSignalMass.push_back(wg2); 

  std::vector<PdfBase*> compsSignalMass;
  compsSignalMass.push_back(gauss1);
  compsSignalMass.push_back(gauss2);
//  compsSignalMass.push_back(gauss3);

  AddPdf signalMass("signalMass", weightsSignalMass, compsSignalMass); 
//  signalMass.addSpecialMask(PdfBase::ForceCommonNorm) ;
  

  vector<Variable*> weightsPoly;
  weightsPoly.push_back(constaCoef);
//  weightsPoly.push_back(linearCoef);
//  weightsPoly.push_back(secondCoef);
//  weightsPoly.push_back(thirdCoef);

  
//  PolynomialPdf* polyTmp = new PolynomialPdf("polyTmp", xMass, weightsPoly); 
//   std::vector<PdfBase*> compsPoly2;
//   compsSignalMass.push_back(polyTmp);
//   compsSignalMass.push_back(polyTmp);
//   
//   ProdPdf* poly      = new ProdPdf("poly"  ,compsPoly2 );
 
  PolynomialPdf* poly = new PolynomialPdf("poly", xMass, weightsPoly); 
  
  std::vector<Variable*> weightsBckgMass;
  weightsBckgMass.push_back(wb1);
  weightsBckgMass.push_back(wb2);
//weightsBckgMass.push_back(wb3);

//  ArgusPdf* argus = new  ArgusPdf("argus", xMass, treshold, aslope, true, apower);  

  std::vector<PdfBase*> compsBckgMass;
  compsBckgMass.push_back(gaussBckgBp);
  compsBckgMass.push_back(gaussBckgB0);
//    compsBckgMass.push_back(argus);
//  compsBckgMass.push_back(poly);
  compsBckgMass.push_back(SimpleCheby2);
 
  AddPdf bckgMass("bckgMass", weightsBckgMass, compsBckgMass);
//  bckgMass.addSpecialMask(PdfBase::ForceCommonNorm) ;

  
//==============================================================================
//==============================================================================
//==============================================================================
// Lifetime
//==============================================================================
//==============================================================================
//==============================================================================

     Variable* cTau     = new Variable("cTau"  ,1./(1.638 *c_const), 10., 40.);
//     Variable* tauSB1   = new Variable("tauSB1",1./(1.620 *c_const), 0., 1000.);
//     Variable* tauSB2   = new Variable("tauSB2",1./(0.400 *c_const), 0., 1000.);

     Variable* tauSB1   = new Variable("tauSB1",1/3.11894e-03, 0., 2000.);
     Variable* tauSB2   = new Variable("tauSB2",1/4.69903e-02, 0., 1000.);
     Variable* tauSB3   = new Variable("tauSB3",1/5.42673e-03, 0., 1000.);

//Variable* cTau     = new Variable("cTau"  ,1./( 1.638 *c_const),0.00, 1000.);
//Variable* tauSB1   = new Variable("tauSB1",1./( 1.440 *c_const),0.00, 1000.);
//Variable* tauSB2   = new Variable("tauSB2",1./( 1.600 *c_const),0.00, 1000.);

  Variable* meanResSign    = new Variable("meanResSign",0);

  Variable* meanResBckg1 = new Variable("meanResBckg1" ,0.);
  Variable* meanResBckg2 = new Variable("meanResBckg2" ,0.);
  Variable* meanResBckg3 = new Variable("meanResBckg3" ,0.);

  Variable* sigmaRes     = new Variable("sigmaRes"      ,0.0003,0,001);

  Variable* sigmaResBckg = new Variable("sigmaResBckg"  ,0.0003,0,001);



//  Variable* meanLandauErrSign      = new Variable( "meanLandauErrSign"        ,0.0015 ,SXMin, SXMax);
//  Variable* sigmaLandauErrorSign   = new Variable( "sigmaLandauErrorSign"     ,0.0002 ,SXMin, SXMax);

//  Variable* meanLandauErrBckg      = new Variable( "meanLandauErrBckg"        ,0.0015 ,SXMin, SXMax);
//  Variable* sigmaLandauErrorBckg   = new Variable( "sigmaLandauErrorBckg"     ,0.0002 ,SXMin, SXMax);
 
//  Variable* meanGaussianErrSign    = new Variable( "meanGaussianErrSign"      ,0.0013  ,SXMin, SXMax);
//  Variable* sigmaGaussianErrorSign = new Variable( "sigmaGaussianErrorSign"   ,0.0003  ,0.00001, SXMax);

//  Variable* meanGaussianErrBckg1   = new Variable( "meanGaussianErrBckg1"      ,0.0013  ,0.,SXMin, SXMax);
//  Variable* sigmaGaussianErrorBckg1= new Variable( "sigmaGaussianErrorBckg1"   ,0.0003  ,0.,0.00001, SXMax);

//  Variable* meanBifurGErrSign      = new Variable( "meanBifurGErrSign"        ,0.0015 ,0., SXMax);
//  Variable* sigmaLBifurGErrSign    = new Variable( "sigmaLBifurGErrSign"      ,0.0003 ,0.00001, SXMax);
//  Variable* sigmaRBifurGErrSign    = new Variable( "sigmaRBifurGErrSign"      ,0.0009 ,0.00001, SXMax);

//  Variable* meanBifurGErrBckg      = new Variable( "meanBifurGErrBckg"	      ,0.0015 ,0., SXMax);
//  Variable* sigmaLBifurGErrBckg    = new Variable( "sigmaLBifurGErrBckg"      ,0.0003 ,0.00001, SXMax);
//  Variable* sigmaRBifurGErrBckg    = new Variable( "sigmaRBifurGErrBckg"      ,0.0009 ,0.00001, SXMax);
  
//   Variable* tauErrSign   = new Variable("tauErrSign",2100,0.,0., 10000.);
//   Variable* tauErrBckg1  = new Variable("tauErrBckg1",2100,0.,0., 10000.);

  Variable* meanGaussianErrSign1    = new Variable( "meanGaussianErrSign1"     ,1.72100e-03  ,0.0001, SXMax);
  Variable* meanGaussianErrSign2    = new Variable( "meanGaussianErrSign2"     ,1.72100e-03  ,0.0001, SXMax);
  Variable* meanGaussianErrSign3    = new Variable( "meanGaussianErrSign3"     ,1.72100e-03  ,0.0001, SXMax);

  Variable* sigmaGaussianErrorSign1 = new Variable( "sigmaGaussianErrorSign1"  ,4.55213e-04  ,0.0001, 0.001);
  Variable* sigmaGaussianErrorSign2 = new Variable( "sigmaGaussianErrorSign2"  ,4.55213e-04  ,0.0001, 0.001);
  Variable* sigmaGaussianErrorSign3 = new Variable( "sigmaGaussianErrorSign3"  ,4.55213e-04  ,0.0001, 0.001);

  Variable* meanGaussianErrBckg1    = new Variable( "meanGaussianErrBckg1"      ,2.00197e-03 ,0.0001, SXMax);
  Variable* meanGaussianErrBckg2    = new Variable( "meanGaussianErrBckg2"      ,1.51338e-03 ,0.00005, 0.005);
  Variable* meanGaussianErrBckg3    = new Variable( "meanGaussianErrBckg3"      ,1.8-03      ,0.0001, SXMax);

  Variable* sigmaGaussianErrorBckg1 = new Variable( "sigmaGaussianErrorBckg1"   ,4.28955e-04,0.0001, 0.001);
  Variable* sigmaGaussianErrorBckg2 = new Variable( "sigmaGaussianErrorBckg2"   ,3.67023e-04,0.0001, 0.001);
  Variable* sigmaGaussianErrorBckg3 = new Variable( "sigmaGaussianErrorBckg3"   , 2.8e-04   ,0.0001, 0.001);


  Variable* tauErrSign1  = new Variable("tauErrSign1" ,1.49315e+03,1000., 20000.);
  Variable* tauErrSign2  = new Variable("tauErrSign2" ,1.49315e+03,1000., 11000.);

  Variable* tauErrBckg1  = new Variable("tauErrBckg1",1/3.73273e-04, 2000., 10000);
  Variable* tauErrBckg2  = new Variable("tauErrBckg2",1/3.72029e-04, 2000., 18000.);
  Variable* tauErrBckg3  = new Variable("tauErrBckg3",1/3.71851e-04, 2000., 20000.);

  Variable* wt1 = new Variable("wt1",0.03, 0., 1.);
  Variable* wt2 = new Variable("wt2",0.05, 0., 1.);
//GenTrue break 5 sigma 0.006-0.35
         double ef0Param = 1.66895e-01       ;
         double ef1Param = 3.02444e+02       ;
         double ef2Param =-1.34418e-01       ;
         double ef3Param =-1.88642e-01       ;
         double ef4Param = 2.93237e+00       ;
         double ef5Param =-2.28278e+01       ;
         double ef6Param = 6.77957e+01       ;
         double ef7Param =-6.83260e+01       ;


//  Variable* ef0 = new Variable("ef0",  5.06489e-02); 
//  Variable* uno = new Variable("uno",  1); 
  Variable* ef0 = new Variable("ef0",  ef0Param); 
  Variable* ef1 = new Variable("ef1",  ef1Param); 
  Variable* ef2 = new Variable("ef2",  ef2Param); 
  Variable* ef3 = new Variable("ef3",  ef3Param); 
  Variable* ef4 = new Variable("ef4",  ef4Param); 
  Variable* ef5 = new Variable("ef5",  ef5Param); 
  Variable* ef6 = new Variable("ef6",  ef6Param); 
  Variable* ef7 = new Variable("ef7",  ef7Param); 

//  coeffEffi.push_back(uno);
//   vector<Variable*> coeffEffi;
//   coeffEffi.push_back(ef0);
//   coeffEffi.push_back(ef1);
//   coeffEffi.push_back(ef2);
//  coeffEffi.push_back(ef3);

 
//   PolyEffiPdf* Effi = new PolyEffiPdf("Effi", xcTau, coeffEffi);
// PolynomialPdf* Effi = new PolynomialPdf("Effi", xcTau, coeffEffi);

//  ErfcPolyPdf  *Effi = new ErfcPolyPdf("Effi", xcTau, ef0,ef1,ef2,ef3,ef4);
      SigmoidBpPdf  *Effi = new SigmoidBpPdf("Effi", xcTau, ef0,ef1,ef2,ef3,ef4,ef5,ef6,ef7);
//      SigmoidGausPdf  *Effi = new SigmoidGausPdf("Effi", xcTau, ef0,ef1,ef2,ef3,ef4,ef5,ef6);


     Variable* XMinV     = new Variable("XMinV"  ,XMin ,0,0,1);
     Variable* XMaxV     = new Variable("XMaxV"  ,XMax ,0,0,1);
     Variable* SXMinV    = new Variable("SXMinV" ,SXMin,0,0,1);
     Variable* SXMaxV    = new Variable("SXMaxV" ,SXMax,0,0,1);
//  Effi->setParameterConstantness(true); 
//  pdfFitBckg1 ->addSpecialMask(PdfBase::ForceSeparateNorm); 
//  Effi        ->addSpecialMask(PdfBase::ForceSeparateNorm); 
   
//    ExpGausProdBPdf* DecayBp	= new ExpGausProdBPdf("DecayBp"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign, meanGaussianErrSign,tauErrSign1,
//    XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* DecayBp1	= new ExpGausProdBPdf("DecayBp1"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign1, meanGaussianErrSign1,tauErrSign2,
      XMinV,XMaxV,SXMinV,SXMaxV);
//  ExpGausProdEffiBPdf* DecayBp1	= new ExpGausProdEffiBPdf("DecayBp1"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign, meanGaussianErrSign,tauErrSign1,
//  XMinV,XMaxV,SXMinV,SXMaxV);
//     ExpGausProdBPdf* DecayBp2	= new ExpGausProdBPdf("DecayBp2"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign2, meanGaussianErrSign2,tauErrSign2,
//     XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* DecayBp2	= new ExpGausProdBPdf("DecayBp2"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign2, meanGaussianErrSign2,tauErrSign2,
      XMinV,XMaxV,SXMinV,SXMaxV);
      ExpGausProdBPdf* DecayBp3 = new ExpGausProdBPdf("DecayBp3"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign3, meanGaussianErrSign3,tauErrSign2,
      XMinV,XMaxV,SXMinV,SXMaxV);
//  ExpGausProdEffiBPdf* DecayBp2	= new ExpGausProdEffiBPdf("DecayBp2"    , xcTau, xScTau, meanResSign   , cTau  , sigmaGaussianErrorSign2, meanGaussianErrSign2,tauErrSign2,
//  XMinV,XMaxV,SXMinV,SXMaxV);
//  ExpGausProdEffiBPdf* pdfFitBckg1 = new ExpGausProdEffiBPdf("pdfFitBckg1", xcTau, xScTau, meanResBckg, tauSB1, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
//  XMinV,XMaxV,SXMinV,SXMaxV);
//        ExpGausProdBPdf* pdfFitBckg1 = new ExpGausProdBPdf("pdfFitBckg1", xcTau, xScTau, meanResBckg, tauSB1, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
//        XMinV,XMaxV,SXMinV,SXMaxV);
       ExpGausProdBPdf* pdfFitBckg1 = new ExpGausProdBPdf("pdfFitBckg1", xcTau, xScTau, meanResBckg2, tauSB1, sigmaGaussianErrorBckg2, meanGaussianErrBckg2,tauErrBckg2,
       XMinV,XMaxV,SXMinV,SXMaxV);
       ExpGausProdBPdf* pdfFitBckg2 = new ExpGausProdBPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg2, tauSB2, sigmaGaussianErrorBckg2, meanGaussianErrBckg2,tauErrBckg2,
       XMinV,XMaxV,SXMinV,SXMaxV);
       ExpGausProdBPdf* pdfFitBckg3 = new ExpGausProdBPdf("pdfFitBckg3", xcTau, xScTau, meanResBckg2, tauSB3, sigmaGaussianErrorBckg2, meanGaussianErrBckg2,tauErrBckg3,
       XMinV,XMaxV,SXMinV,SXMaxV);
//        ExpGausProdBPdf* pdfFitBckg3 = new ExpGausProdBPdf("pdfFitBckg3", xcTau, xScTau, meanResBckg3, tauSB3, sigmaGaussianErrorBckg3, meanGaussianErrBckg3,tauErrBckg3,
//        XMinV,XMaxV,SXMinV,SXMaxV);
 // ExpGausProdEffiBPdf* pdfFitBckg2 = new ExpGausProdEffiBPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg, tauSB2, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg1,
 // XMinV,XMaxV,SXMinV,SXMaxV);
//       ExpGausProdBPdf* pdfFitBckg2 = new ExpGausProdBPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg, tauSB2, sigmaGaussianErrorBckg2, meanGaussianErrBckg2,tauErrBckg2,
//       XMinV,XMaxV,SXMinV,SXMaxV);
//  ExpGausProdBPdf* pdfFitBckg2 = new ExpGausProdBPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg, tauSB2, sigmaGaussianErrorBckg1, meanGaussianErrBckg1,tauErrBckg2,
//  XMinV,XMaxV,SXMinV,SXMaxV);

// ExpPdf* DecayBp     = new ExpPdf("DecayBp"    ,  xcTau, cTau  );
// ExpPdf* pdfFitBckg1 = new ExpPdf("pdfFitBckg1", xcTau, tauSB1);
// ExpPdf* pdfFitBckg2 = new ExpPdf("pdfFitBckg2", xcTau, tauSB2);


//  ExpGausPEEPdf* DecayBp     = new ExpGausPEEPdf("DecayBp"    , xcTau, xScTau, meanResSign   , cTau  );
//  ExpGausPEEPdf* pdfFitBckg1 = new ExpGausPEEPdf("pdfFitBckg1", xcTau, xScTau, meanResBckg, tauSB1);
//  ExpGausPEEPdf* pdfFitBckg2 = new ExpGausPEEPdf("pdfFitBckg2", xcTau, xScTau, meanResBckg, tauSB2);

  
//  ExpGausPdf* DecayBp	    = new ExpGausPdf("DecayBp"	, xcTau, meanResSign   , sigmaRes, cTau  );
//  ExpGausPdf* pdfFitBckg1 = new ExpGausPdf("pdfFitBckg1", xcTau, meanResBckg, sigmaResBckg, tauSB1);
//  ExpGausPdf* pdfFitBckg2 = new ExpGausPdf("pdfFitBckg2", xcTau, meanResBckg, sigmaResBckg, tauSB2);

//    std::vector<PdfBase*> compspdfFitBp1;
//    compspdfFitBp1.push_back(DecayBp1);
//    compspdfFitBp1.push_back(Effi);
//    ProdPdf* pdfFitBp1	 = new ProdPdf("pdfFitBp1"  , compspdfFitBp1);
// 
//    std::vector<PdfBase*> compspdfFitBp2;
//    compspdfFitBp2.push_back(DecayBp2);
//    compspdfFitBp2.push_back(Effi);
//    ProdPdf* pdfFitBp2	 = new ProdPdf("pdfFitBp2"  , compspdfFitBp2);
// 
//   std::vector<PdfBase*> compspdfDecayBpAdd;
//   compspdfDecayBpAdd.push_back(pdfFitBp1);
//   compspdfDecayBpAdd.push_back(pdfFitBp2);
//   AddPdf *pdfFitBp = new AddPdf("pdfFitBp", weightsSignalMass, compspdfDecayBpAdd);
// 
   std::vector<Variable*> weightsSignalTau;
   weightsSignalTau.push_back(wt1);
//   weightsSignalTau.push_back(wt2); 
   std::vector<PdfBase*> compspdfDecayBpAdd;
   compspdfDecayBpAdd.push_back(DecayBp1);
   compspdfDecayBpAdd.push_back(DecayBp2);
//   compspdfDecayBpAdd.push_back(DecayBp3);
   AddPdf DecayBp("DecayBp", weightsSignalMass, compspdfDecayBpAdd);
//   AddPdf DecayBp("DecayBp", weightsSignalTau, compspdfDecayBpAdd);
//  AddPdf* pdfFitBp =  new AddPdf("pdfFitBp", weightsSignalMass, compspdfDecayBpAdd);
  
 
//  DecayBp.addSpecialMask(PdfBase::ForceSeparateNorm); 
 
 
 
    std::vector<PdfBase*> compspdfFitBp;
    compspdfFitBp.push_back(&DecayBp);
//    compspdfFitBp.push_back(DecayBp1);
    compspdfFitBp.push_back(Effi);
    ProdPdf* pdfFitBp	 = new ProdPdf("pdfFitBp"  , compspdfFitBp);

  Variable* b1 = new Variable("b1",5.38694e-02,0., 1.);
  Variable* b2 = new Variable("b2",8.97145e-01,0., 1.);
  std::vector<Variable*> weightspdfFitBckg;
//  weightspdfFitBckg.push_back(b1);
  weightspdfFitBckg.push_back(b2);

  
  std::vector<PdfBase*> compspdfFitBckgAdd;
//  compspdfFitBckgAdd.push_back(pdfFitBckg1);
  compspdfFitBckgAdd.push_back(pdfFitBckg2);
  compspdfFitBckgAdd.push_back(pdfFitBckg3);
//  AddPdf pdfFitBckgAdd("pdfFitBckgAdd", weightspdfFitBckg, compspdfFitBckgAdd);
//  AddPdf pdfFitBckgTmp("pdfFitBckgTmp",weightsBckgMass , compspdfFitBckgAdd);
  AddPdf pdfFitBckgTmp("pdfFitBckgTmp", weightspdfFitBckg, compspdfFitBckgAdd);
//      AddPdf pdfFitBckg("pdfFitBckg", weightspdfFitBckg, compspdfFitBckgAdd);
  
   std::vector<PdfBase*> compspdfFitBpBckg;
   compspdfFitBpBckg.push_back(&pdfFitBckgTmp);
//   compspdfFitBpBckg.push_back(pdfFitBckg1);
   compspdfFitBpBckg.push_back(Effi);
   ProdPdf pdfFitBckg("pdfFitBckg"  , compspdfFitBpBckg);
 

///////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////
//
// 2DFit
//  
///////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////
  

  std::vector<PdfBase*> compsSignalLife;
  compsSignalLife.push_back(&signalMass);
  compsSignalLife.push_back(pdfFitBp);
//  compsSignalLife.push_back( GaussianErrorSign);
  //compsSignalLife.push_back(ExpGauSign);
//  compsSignalLife.push_back( LandauErrorSign);
  ///compsSignalLife.push_back(BifurGErrorSign);
//  compsSignalLife.push_back(DecayBp);

  std::vector<PdfBase*> compsBckgLife;
  compsBckgLife.push_back(&bckgMass);
  compsBckgLife.push_back(&pdfFitBckg);
//  compsBckgLife.push_back(pdfFitBckg1);
//  compsSignalLife.push_back( GaussianErrorBckg);
  //compsSignalLife.push_back(ExpGauBckg);
//  compsSignalLife.push_back( LandauErrorBckg);
  //compsSignalLife.push_back(BifurGErrorBckg);
//compsBckgLife.push_back(&pdfFitBckgAdd);

  ProdPdf* signalLife = new ProdPdf("signalLife", compsSignalLife);
  ProdPdf* bckgLife   = new ProdPdf("bckgLife  ", compsBckgLife);

  std::vector<Variable*> weightsYield;
  weightsYield.push_back(signalYield);
  weightsYield.push_back(bckgYield);
  

  std::vector<PdfBase*> compsModel;
  
  
  compsModel.push_back(signalLife);
//  compsModel.push_back(gaussBckgBp);
  compsModel.push_back(bckgLife);
  
  
//  compsModel.push_back(poly);
//  compsModel.push_back(argus);
  AddPdf model("model", weightsYield, compsModel); 

//  model.addSpecialMask(PdfBase::ForceCommonNorm) ;

//
// These are used for Plots....
//
  std::vector<PdfBase*> compsMass;
  compsMass.push_back(&signalMass);
  compsMass.push_back(&bckgMass);
  AddPdf modelMass("modelMass", weightsYield, compsMass); 
  
  std::vector<PdfBase*> compscTau;
  compscTau.push_back(pdfFitBp);
  compscTau.push_back(&pdfFitBckg);
  AddPdf model_cTau("model_cTau", weightsYield, compscTau); 
  
//  std::vector<PdfBase*> compsSTau;
//  compsSTau.push_back(LandauErrorSign);
//  compsSTau.push_back(LandauErrorBckg);
//  compsSTau.push_back(BifurGErrorSign);
//  compsSTau.push_back(BifurGErrorBckg);
//  compsSTau.push_back(GaussianErrorSign);
//  compsSTau.push_back(GaussianErrorBckg);
//  compsSTau.push_back(ExpGauSign);
//  compsSTau.push_back(ExpGauBckg);
//  AddPdf model_STau("model_cTau", weightsYield, compsSTau); 
  
  

//
// Data
//
  vector<Variable*> dataVec;
  
  dataVec.push_back(xMass);
  dataVec.push_back(xcTau);
  dataVec.push_back(xScTau);
  UnbinnedDataSet* dataLife = new UnbinnedDataSet(dataVec);
//
  if (!InputFile)
   {
     cout<<"File:"<<InputFileName<<" not found!!!"<<endl;
    exit(1);
   }
   InputFile->ls();
   
   TTree *TauBpTree    = (TTree*)InputFile->Get(InputTauBpTreeName);
   if(!TauBpTree ){
     cout<<"TTree cTau Data: "<< InputTauBpTreeName <<" not found!!!"<<endl;
     exit(1);
   }else{
     cout<<"TTree cTau Data: "<< InputTauBpTreeName <<" OK FOUND!!!"<<endl;
   }  
    
   TauBpTree->SetBranchAddress("xBpMass",&xBpMass);
//   TauBpTree->SetBranchAddress("xBpTau" ,&xBpTau);
   TauBpTree->SetBranchAddress("xBpcTau",&xBpcTau);
   TauBpTree->SetBranchAddress("xSBpcTau",&xSBpcTau);
   int nentries = (int)TauBpTree->GetEntries();
   
   for (Int_t i=0;i<nentries;i++) { 
    TauBpTree->GetEntry(i);
    if(xBpcTau>XMin&&xBpcTau<XMax&&xSBpcTau>SXMin&&xSBpcTau<SXMax){
     if(xBpMass>XMinSign&&xBpMass<XMaxSign){
      xMass->value = xBpMass;
      xcTau->value = xBpcTau;
      xScTau->value = xSBpcTau; 
      dataLife->addEvent();
      HxMass.Fill(xBpMass);
      HxcTau.Fill(xBpcTau);
      HxScTau.Fill(xSBpcTau);
     } 
     if(xBpMass>XMinSBL&&xBpMass<XMaxSBL){
      HxcTauSB.Fill(xBpcTau);
      HxScTauSB.Fill(xSBpcTau);
     } 
     if(xBpMass>XMinSBR&&xBpMass<XMaxSBR){
      HxcTauSB.Fill(xBpcTau);
      HxScTauSB.Fill(xSBpcTau);
     }  
    } 
   }
   cout<<"***********************************"<<endl;
   cout<<"***********************************\n"<<endl;
   cout<<"TauBpTree   Entries = "<<nentries<<endl;
   cout<<"Mass        Entries = "<<HxMass.GetEntries()<<endl;
   cout<<"SideBand    Entries = "<<HxcTauSB.GetEntries()<<endl;
   cout<<"\n***********************************"<<endl;
   cout<<"***********************************"<<endl;
//================================================================================
//================================================================================
///FIT
//================================================================================
//================================================================================
//  double arglis[10];
//  int ierflg= 0;
  model.setData(dataLife);
  FitManager fitter(&model);
  fitter.setMaxCalls(8000);
  cout<<"                  ===*** Start Fit ***=== "<<endl;
  cout<<"                  ===*** Start Fit ***=== "<<endl;
  cout<<"                  ===*** Start Fit ***=== "<<endl;
  fitter.setupMinuit();
  TMinuit * Minuit = fitter.getMinuitObject();  
// 
//       Minuit->FixParameter(7);
//      fitter.runCommand("MIGRAD");
   
//      Minuit->Release(7);
   fitter.runCommand("MIGRAD");
 
//  fitter.runCommand("MINOS");
   fitter.runCommand("HESSE");
//  fitter.fit();   
//  Minuit->SetPrintLevel(1);
//  Minuit->mnmigr();
//  Minuit->mnhess();
//  Minuit->mnmigr();
  fitter.getMinuitValues(); 
  cout<<"		   ===***  End  Fit ***=== "<<endl;
  cout<<"		   ===***  End  Fit ***=== "<<endl;
  cout<<"		   ===***  End  Fit ***=== "<<endl;
  
//================================================================================
//================================================================================
///FIT
//================================================================================
//================================================================================

//================================================================================
///PLOT

//XHScale=10;
// Mass
  UnbinnedDataSet gridMass(xMass);
  double totalDataMass = 0; 
  double NStepMass = XHScale*xMass->numbins;
  for (int i = 0; i < NStepMass; ++i) {
    double step = (xMass->upperlimit - xMass->lowerlimit)/NStepMass;
    xMass->value = xMass->lowerlimit + (i + 0.5) * step;
    gridMass.addEvent(); 
   totalDataMass++; 
  }

  modelMass.setData(&gridMass);
  vector<vector<double> > pdfValsMass;
  modelMass.getCompProbsAtDataPoints(pdfValsMass); 
  double totalPdfMass = 0; 
  for (int i = 0; i < gridMass.getNumEvents(); ++i) {
    gridMass.loadEvent(i); 
    pdfHist.Fill(xMass->value, pdfValsMass[0][i]);
    sigHist.Fill(xMass->value, pdfValsMass[1][i]);
    bkgHist.Fill(xMass->value, pdfValsMass[2][i]);
    totalPdfMass += pdfValsMass[0][i]; 
  }
  
  
  pdfHist.Scale((signalYield->value+bckgYield->value)/pdfHist.Integral()*XHScale);
  sigHist.Scale(signalYield->value/sigHist.Integral()*XHScale);
  bkgHist.Scale(bckgYield->value/bkgHist.Integral()*XHScale);
  std::cout<<"Signal Yield = "<< signalYield->value<<std::endl;
  std::cout<<"Bckg   Yield = "<< bckgYield->value<<std::endl;
  std::cout<<"(SB    Yield  = "<<HxcTauSB.GetEntries() <<")"<<std::endl;
  std::cout<<"Tot   Yield  = "<< signalYield->value+bckgYield->value<<std::endl;
//--------------------------------------------------  
// Tau


//XHScale=1;
  int NIntegral = 1;
  

//  vector<Variable*> dataPlot;
//  dataPlot.push_back(xcTau);
//  dataPlot.push_back(xScTau);

  vector<Variable*> dataPlot2D;
  dataPlot2D.push_back(xcTau);
  dataPlot2D.push_back(xScTau);

//  vector<Variable*> dataPlotS;
//  dataPlotS.push_back(xScTau);
  
  
//  UnbinnedDataSet grid_cTau(dataPlot);
  UnbinnedDataSet grid_cTau2D(dataPlot2D);
//  UnbinnedDataSet grid_STau(dataPlotS);
  
//  bool first = true;
//  UnbinnedDataSet grid_cTau(xcTau);
//  double totalData_cTau = 0; 
  double NStep  = XHScale*xcTau->numbins;
//  double NSStep   = XHScale*xcTau->numbins;
  double NSStep2D = NIntegral*XHScale*xScTau->numbins;
  double step  = (xcTau->upperlimit - xcTau->lowerlimit)/NStep;
//  double sstep = (xScTau->upperlimit - xScTau->lowerlimit)/NSStep;
  double sstep2D = (xScTau->upperlimit - xScTau->lowerlimit)/NSStep2D;
  for (int i = 0; i < NStep; ++i) {
    xcTau->value  = xcTau->lowerlimit  + (i + 0.5) * step;
//    grid_cTau.addEvent(); 
//    totalData_cTau++; 
//    xScTau->value = xScTau->lowerlimit + (i + 0.5) * sstep;
//    cout<<"X = "<<xcTau->value<<" sx = "<<xScTau->value<<endl;
//    grid_cTau.addEvent(); 
//    xcTau2D->value  = xcTau2D ->lowerlimit + (i + 0.5) * step;
//   cout<<"======================================     \n"<<endl;
//   cout<<"======================================     \n"<<endl;
//   cout<<"======================================     \n"<<endl;
    for (int ii = 0; ii < NSStep2D; ++ii) {
     xScTau->value = xScTau->lowerlimit + (ii + 0.5) * sstep2D;
//     xScTau2D->value = xScTau2D->lowerlimit + (ii + 0.5) * sstep2D;
//    cout<<"X = "<<xcTau->value<<" sx = "<<xScTau->value<<endl;
     grid_cTau2D.addEvent(); 
//     if (first) grid_STau.addEvent();
    }
//    first = false;
  }

//  model_cTau.setData(&grid_cTau);
//  vector<vector<double> > pdfVals_cTau;
//  model_cTau.getCompProbsAtDataPoints(pdfVals_cTau); 
//  double totalPdf_cTau = 0; 
//   for (int i = 0; i < grid_cTau.getNumEvents(); ++i) {
//     grid_cTau.loadEvent(i); 
//     pdf_cTau_Hist.Fill(xcTau->value , pdfVals_cTau[0][i]);
//     sig_cTau_Hist.Fill(xcTau->value , pdfVals_cTau[1][i]);
//     bkg_cTau_Hist.Fill(xcTau->value , pdfVals_cTau[2][i]);
//     totalPdf_cTau += pdfVals_cTau[0][i]; 
//   }

//  double pdf_cTau_Integral2D = 0;
//  double sig_cTau_Integral2D = 0;
//  double bkg_cTau_Integral2D = 0;
//model.setData(&grid_cTau2D);
//vector<vector<double> > pdfVals_cTau2D;
//model.getCompProbsAtDataPoints(pdfVals_cTau2D);
     model_cTau.setData(&grid_cTau2D);
     vector<vector<double> > pdfVals_cTau2D;
     model_cTau.getCompProbsAtDataPoints(pdfVals_cTau2D);
  for (int i = 0; i < grid_cTau2D.getNumEvents(); ++i) {
    grid_cTau2D.loadEvent(i); 
    pdf_cTauSTau_Hist2D.Fill(xcTau->value ,xScTau->value , pdfVals_cTau2D[0][i]);
    sig_cTauSTau_Hist2D.Fill(xcTau->value ,xScTau->value , pdfVals_cTau2D[1][i]);
    bkg_cTauSTau_Hist2D.Fill(xcTau->value ,xScTau->value , pdfVals_cTau2D[2][i]);
//     if (i%int(NSStep2D) == 1 && i>0){
//      pdf_cTau_Hist2D.Fill(xcTau->value , pdf_cTau_Integral2D/step);
//      sig_cTau_Hist2D.Fill(xcTau->value , sig_cTau_Integral2D/step);
//      bkg_cTau_Hist2D.Fill(xcTau->value , bkg_cTau_Integral2D/step);
// //     cout<<"Int = "<<bkg_cTau_Integral2D<<endl;
//      pdf_cTau_Integral2D=0;
//      sig_cTau_Integral2D=0;
//      bkg_cTau_Integral2D=0;
// //     exit(0);
//     }else{
// //     cout<<"X = "<<xcTau->value<<" sx = "<<xScTau->value<<" NStep2D = "<<NStep2D<<endl;
//      pdf_cTau_Integral2D =+ pdfVals_cTau2D[0][i]*sstep2D; 
//      sig_cTau_Integral2D =+ pdfVals_cTau2D[1][i]*sstep2D; 
//      bkg_cTau_Integral2D =+ pdfVals_cTau2D[2][i]*sstep2D;
// //     cout<<"Int = "<<pdfVals_cTau2D[2][i]<<endl;
//     } 
  }
  
  TH1D * pdf_cTauSTau_X = pdf_cTauSTau_Hist2D.ProjectionX("pdf_cTauSTau_X");
  TH1D * pdf_cTauSTau_Y = pdf_cTauSTau_Hist2D.ProjectionY("pdf_cTauSTau_Y");

  TH1D * sig_cTauSTau_X = sig_cTauSTau_Hist2D.ProjectionX("sig_cTauSTau_X");
  TH1D * sig_cTauSTau_Y = sig_cTauSTau_Hist2D.ProjectionY("sig_cTauSTau_Y");

  TH1D * bkg_cTauSTau_X = bkg_cTauSTau_Hist2D.ProjectionX("bkg_cTauSTau_X");
  TH1D * bkg_cTauSTau_Y = bkg_cTauSTau_Hist2D.ProjectionY("bkg_cTauSTau_Y");

/*   vector<Variable*> dataSPlot;
  dataSPlot.push_back(xScTau);
  dataSPlot.push_back(xcTau);
  UnbinnedDataSet grid_STau(dataSPlot);
//  UnbinnedDataSet grid_cTau(xcTau);
  double totalData_STau = 0; 
  NStep  = XHScale*xScTau->numbins;
  double NSStep = XHScale*xcTau->numbins;
  for (int i = 0; i < NSStep; ++i) {
    totalData_STau++; 
    double sstep = (xScTau->upperlimit - xScTau->lowerlimit)/NSStep;
    xScTau->value = xScTau->lowerlimit + (i + 0.5) * sstep;
    grid_STau.addEvent(); 
  }
 */

//
// STau
//
//   model_STau.setData(&grid_STau);
//   vector<vector<double> > pdfVals_STau;
//   model_STau.getCompProbsAtDataPoints(pdfVals_STau); 
//   double totalPdf_STau = 0;  
//   for (int i = 0; i < grid_STau.getNumEvents(); ++i) {
//     grid_STau.loadEvent(i); 
//     pdf_STau_Hist.Fill(xScTau->value, pdfVals_STau[0][i]);
//     sig_STau_Hist.Fill(xScTau->value, pdfVals_STau[1][i]);
//     bkg_STau_Hist.Fill(xScTau->value, pdfVals_STau[2][i]);
//     totalPdf_STau += pdfVals_STau[0][i]; 
//   }
  
//
// Models plot  
//   pdf_cTau_Hist.Scale((signalYield->value+bckgYield->value)/pdf_cTau_Hist.Integral()*XHScale);
//   sig_cTau_Hist.Scale(signalYield->value/sig_cTau_Hist.Integral()*XHScale);
//   bkg_cTau_Hist.Scale(HxcTauSB.GetEntries()/bkg_cTau_Hist.Integral()*XHScale);
// 
//   pdf_cTau_Hist2D.Scale((signalYield->value+bckgYield->value)/pdf_cTau_Hist2D.Integral()*XHScale);
//   sig_cTau_Hist2D.Scale(signalYield->value/sig_cTau_Hist2D.Integral()*XHScale);
//   bkg_cTau_Hist2D.Scale(HxcTauSB.GetEntries()/bkg_cTau_Hist2D.Integral()*XHScale);
    
  pdf_cTauSTau_X->Scale((signalYield->value+bckgYield->value)/pdf_cTauSTau_X->Integral()*XHScale);
  sig_cTauSTau_X->Scale((signalYield->value)/sig_cTauSTau_X->Integral()*XHScale);
  bkg_cTauSTau_X->Scale((HxcTauSB.GetEntries())/bkg_cTauSTau_X->Integral()*XHScale);
//  bkg_cTauSTau_X->Scale((bckgYield->value)/bkg_cTauSTau_X->Integral()*XHScale);
  pdf_cTauSTau_Y->Scale((signalYield->value+bckgYield->value)/pdf_cTauSTau_Y->Integral()*XHScale);
  sig_cTauSTau_Y->Scale((signalYield->value)/sig_cTauSTau_Y->Integral()*XHScale);
  bkg_cTauSTau_Y->Scale((HxcTauSB.GetEntries())/bkg_cTauSTau_Y->Integral()*XHScale);
//  bkg_cTauSTau_Y->Scale((bckgYield->value)/bkg_cTauSTau_Y->Integral()*XHScale);
  
  

//   sig_STau_Hist.Scale(signalYield->value/sig_STau_Hist.Integral()*XHScale);
//   bkg_STau_Hist.Scale(HxScTauSB.GetEntries()/bkg_STau_Hist.Integral()*XHScale);
//   pdf_STau_Hist.Scale((signalYield->value+bckgYield->value)/pdf_STau_Hist.Integral()*XHScale);
//  bkg_cTau_Hist.Scale(bckgYield->value/bkg_cTau_Hist.Integral()*XHScale);
 
//   for (int i = 0; i < xMass->numbins; ++i) {
//     double val = pdfHist.GetBinContent(i+1); 
//     val /= totalPdf; 
//     val *= totalData;
//     pdfHist.SetBinContent(i+1, val); 
//     val = sigHist.GetBinContent(i+1); 
//     val /= totalPdf; 
//     val *= sigFrac->value; 
//     val *= totalData;
//     sigHist.SetBinContent(i+1, val); 
//     val = bkgHist.GetBinContent(i+1); 
//     val /= totalPdf; 
//     val *= (1.0 - sigFrac->value);
//     val *= totalData;
//     bkgHist.SetBinContent(i+1, val); 
//   }

  double sigmaw    = sigma1->value*wg1->value+ (1-wg1->value)*sigma2->value;
  double sigmawErr = sqrt(sigma1->error*wg1->value*sigma1->error*wg1->value+ (1-wg1->value)*sigma2->error*(1-wg1->value)*sigma2->error);
  c1->cd();
    TLegend* leg_sign = new TLegend(0.30,0.75,0.90,0.90);
    leg_sign->SetTextSize(0.025) ;
    leg_sign->SetTextAlign(31);
    leg_sign->SetBorderSize(0.);
    leg_sign->SetFillStyle(0);
    leg_sign->SetHeader("B^{+} mass spectrum     ");
    if(signalYield->error!=0){
      leg_sign->AddEntry(&HxMass ,Form( "Yield_{Sign} =    %5.0f  #pm %5.0f",signalYield->value,signalYield->error),"");
    }else{
      leg_sign->AddEntry(&HxMass ,Form( "Yield_{Sign} =    %5.0f Fixed",signalYield->value),"");
    }
    if(bckgYield->error!=0){
      leg_sign->AddEntry(&HxMass ,Form( "Yield_{Bckg} =   %5.0f  #pm %5.0f",bckgYield->value,bckgYield->error),"");
    }else{
      leg_sign->AddEntry(&HxMass ,Form( "Yield_{Bckg} =   %5.0f Fixed",bckgYield->value),"");
    }
    
    if(mean->error!=0){
     leg_sign->AddEntry(&HxMass ,Form( "M_{B^{+}} =   %5.4f  #pm %5.4f",mean->value,mean->error),"");
    }else{
     leg_sign->AddEntry(&HxMass ,Form( "M_{B^{+}} =   %5.4f Fixed",mean->value),"");
     }
    if(sigma1->error!=0){
     leg_sign->AddEntry(&HxMass ,Form( "#sigma_{B^{+}} =   %5.4f  #pm %5.4f",sigmaw,sigmawErr),"");
    }else{
     leg_sign->AddEntry(&HxMass ,Form( "#sigma_{B^{+}} =   %5.4f Fixed",sigmaw),"");
    }
  HxMass.SetMarkerStyle(8);
  HxMass.SetMarkerSize(MarkerSize);
  HxMass.SetTitle("");
  HxMass.Draw("E1"); 
  leg_sign->Draw("same");
//  HxMass.Draw("p"); 
  pdfHist.SetLineColor(kBlue);
  pdfHist.SetLineWidth(PlotLineWidth); 
  pdfHist.Draw("same"); 
  sigHist.SetLineColor(kMagenta);
  sigHist.SetLineStyle(kDashed); 
  sigHist.SetLineWidth(PlotLineWidth); 
  sigHist.Draw("same"); 
  bkgHist.SetLineColor(kRed);
  bkgHist.SetLineStyle(kDashed); 
  bkgHist.SetLineWidth(PlotLineWidth); 
  bkgHist.Draw("same"); 
  HxMass.Write();
  pdfHist.Write();
  sigHist.Write();
  bkgHist.Write();
//  
  c2->cd();
  c2->SetLogy();
  TLegend* leg_pdfSB = new TLegend(0.60,0.65,0.90,0.90);
  leg_pdfSB->SetTextAlign(12);
  leg_pdfSB->SetHeader("B^{+} proper time Fit Projections");
  leg_pdfSB->SetTextSize(0.025) ;
  leg_pdfSB->SetBorderSize(0.);
  leg_pdfSB->SetFillStyle(0);
  leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[4]{#scale[1.5]{#tau}_{B^{+}}  =  %5.3f #pm %5.3f     }",1/(c_const*cTau->value),cTau->error/((c_const*cTau->value)*(cTau->value)))   ,"");
  if( b1->error!=0){
      leg_pdfSB->AddEntry(&HxcTau ,Form( "b1   =  %5.3f #pm %5.3f     ",b1->value,b1->error)   ,"");
  }else{      
      leg_pdfSB->AddEntry(&HxcTau ,Form( "b1   =  %5.3f     Fixed     ",b1->value)   ,"");
  }   
  if( b2->error!=0){
      leg_pdfSB->AddEntry(&HxcTau ,Form( "b2   =  %5.3f #pm %5.3f     ",b2->value,b2->error)   ,"");
  }else{      
      leg_pdfSB->AddEntry(&HxcTau ,Form( "b2   =  %5.3f     Fixed     ",b2->value)   ,"");
  }   
  if( tauSB1->error!=0){
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB1} =  %5.3f #pm %5.3f     }",1/(tauSB1->value),tauSB1->error/((tauSB1->value)*(tauSB1->value)))   ,"");
  }else{      
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB1} =  %5.3f	 Fixed	  }",1/(tauSB1->value))   ,"");
  }   
  if( tauSB2->error!=0){
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB2} =  %5.3f #pm %5.3f     }",1/(tauSB2->value),tauSB2->error/((tauSB2->value)*(tauSB2->value)))   ,"");
  }else{      
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB2} =  %5.3f	 Fixed	  }",1/(tauSB2->value))   ,"");
  }   
  if( tauSB3->error!=0){
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB2} =  %5.3f #pm %5.3f     }",1/(tauSB3->value),tauSB3->error/((tauSB3->value)*(tauSB3->value)))   ,"");
  }else{      
      leg_pdfSB->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{SB2} =  %5.3f Fixed     }",1/(tauSB3->value))   ,"");
  }   
  leg_pdfSB->AddEntry(&HxcTau ,"#color[4]{#scale[0.8]{- Fit model	     }}"   ,"");
  leg_pdfSB->AddEntry(&HxcTau ,"#color[6]{#scale[0.8]{- Signal model	     }}"   ,"");
  leg_pdfSB->AddEntry(&HxcTau ,"#color[2]{#scale[0.8]{- Background model on SB}}"   ,"");
  HxcTau.SetMarkerStyle(8);
  HxcTau.SetMarkerSize(MarkerSize);
  HxcTau.SetTitle("");
  HxcTau.Draw("E1");
  HxcTauSB.SetMarkerStyle(8);
  HxcTauSB.SetMarkerSize(0.5);
  HxcTauSB.SetMarkerColor(kRed);
  HxcTauSB.Draw("same,E1");
  leg_pdfSB->Draw("same");
/*   pdf_cTau_Hist2D.SetLineColor(kBlue);
  pdf_cTau_Hist2D.SetLineWidth(3); 
  pdf_cTau_Hist2D.Draw("same"); 
  sig_cTau_Hist2D.SetLineColor(kMagenta);
  sig_cTau_Hist2D.SetLineStyle(kDashed); 
  sig_cTau_Hist2D.SetLineWidth(2); 
  sig_cTau_Hist2D.Draw("same"); 
  bkg_cTau_Hist2D.SetLineColor(kRed);
  bkg_cTau_Hist2D.SetLineStyle(kDashed); 
  bkg_cTau_Hist2D.SetLineWidth(2); 
  bkg_cTau_Hist2D.Draw("same"); 
 */  
  pdf_cTauSTau_X->SetLineColor(kBlue);
  pdf_cTauSTau_X->SetLineWidth(PlotLineWidth);
  pdf_cTauSTau_X->Draw("same");
  sig_cTauSTau_X->SetLineColor(kMagenta);
  sig_cTauSTau_X->SetLineWidth(PlotLineWidth);
  sig_cTauSTau_X->SetLineStyle(kDashed);
  sig_cTauSTau_X->Draw("same");
  bkg_cTauSTau_X->SetLineColor(kRed);
  bkg_cTauSTau_X->SetLineWidth(PlotLineWidth);
  bkg_cTauSTau_X->SetLineStyle(kDashed);
  bkg_cTauSTau_X->Draw("same");
//   pdf_cTau_Hist.SetLineColor(kBlue);
//   pdf_cTau_Hist.SetLineWidth(3); 
//   pdf_cTau_Hist.Draw("same"); 
//   sig_cTau_Hist.SetLineColor(kMagenta);
//   sig_cTau_Hist.SetLineStyle(kDashed); 
//   sig_cTau_Hist.SetLineWidth(2); 
//   sig_cTau_Hist.Draw("same"); 
//   bkg_cTau_Hist.SetLineColor(kRed);
//   bkg_cTau_Hist.SetLineStyle(kDashed); 
//   bkg_cTau_Hist.SetLineWidth(2); 
//   bkg_cTau_Hist.Draw("same"); 
  
  
  c3->cd();
  TLegend* leg_pdfResolution = new TLegend(0.40,0.65,0.90,0.90);
  leg_pdfResolution->SetTextAlign(12);
  leg_pdfResolution->SetHeader("B^{+} resolution Fit Projections");
  leg_pdfResolution->SetTextSize(0.025) ;
  leg_pdfResolution->SetBorderSize(0.);
  leg_pdfResolution->SetFillStyle(0);
//  leg_pdfResolution->AddEntry(&HxcTau ,Form( "#color[4]{#scale[1.5]{#tau}_{B^{+}}  =  %5.3f #pm %5.3f     }",1/(c_const*cTau->value),cTau->error/((c_const*cTau->value)*(cTau->value)))   ,"");
//   if( b1->error!=0){
//       leg_pdfResolution->AddEntry(&HxcTau ,Form( "b1   =  %5.3f #pm %5.3f     ",b1->value,b1->error)   ,"");
//   }else{      
//       leg_pdfResolution->AddEntry(&HxcTau ,Form( "b1   =  %5.3f     Fixed     ",b1->value)   ,"");
//   }   
//   if( tauResolution1->error!=0){
//       leg_pdfResolution->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{Resolution1} =  %5.3f #pm %5.3f     }",1/(c_const*tauResolution1->value),tauResolution1->error/((c_const*tauResolution1->value)*(tauResolution1->value)))   ,"");
//   }else{      
//       leg_pdfResolution->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{Resolution1} =  %5.3f	Fixed	  }",1/(c_const*tauResolution1->value))   ,"");
//   }   
//   if( tauResolution2->error!=0){
//       leg_pdfResolution->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{Resolution2} =  %5.3f #pm %5.3f     }",1/(c_const*tauResolution2->value),tauResolution2->error/((c_const*tauResolution2->value)*(tauResolution2->value)))   ,"");
//   }else{      
//       leg_pdfResolution->AddEntry(&HxcTau ,Form( "#color[2]{#scale[1.5]{#tau}_{Resolution2} =  %5.3f	Fixed	  }",1/(c_const*tauResolution2->value))   ,"");
//   }   
  leg_pdfResolution->AddEntry(&HxScTau ,"#color[4]{#scale[0.8]{- Pdf  Resolution model	     }}"   ,"");
  leg_pdfResolution->AddEntry(&HxScTau ,"#color[6]{#scale[0.8]{- Signal  Resolution  model   }}"   ,"");
  leg_pdfResolution->AddEntry(&HxScTau ,"#color[2]{#scale[0.8]{- Background Resolution model (on SB events)}}"   ,"");
  HxScTau.Draw("E1");
  leg_pdfResolution->Draw("same");
  HxScTauSB.SetMarkerStyle(8);
  HxScTauSB.SetMarkerSize(MarkerSize);
  HxScTauSB.SetMarkerColor(kRed);
  HxScTauSB.Draw("same,E1");
  pdf_cTauSTau_Y->SetLineColor(kBlue);
  pdf_cTauSTau_Y->SetLineWidth(PlotLineWidth);
  pdf_cTauSTau_Y->Draw("same");
  sig_cTauSTau_Y->SetLineColor(kMagenta);
  sig_cTauSTau_Y->SetLineWidth(PlotLineWidth);
  sig_cTauSTau_Y->SetLineStyle(kDashed);
  sig_cTauSTau_Y->Draw("same");
  bkg_cTauSTau_Y->SetLineColor(kRed);
  bkg_cTauSTau_Y->SetLineWidth(PlotLineWidth);
  bkg_cTauSTau_Y->SetLineStyle(kDashed);
  bkg_cTauSTau_Y->Draw("same");


//   pdf_STau_Hist.SetLineColor(kBlue);
//   pdf_STau_Hist.SetLineWidth(2); 
//   pdf_STau_Hist.Draw("same"); 
//   sig_STau_Hist.SetLineColor(kMagenta);
//   sig_STau_Hist.SetLineStyle(kDashed); 
//   sig_STau_Hist.SetLineWidth(2); 
//   sig_STau_Hist.Draw("same"); 
//   bkg_STau_Hist.SetLineColor(kRed);
//   bkg_STau_Hist.SetLineStyle(kDashed); 
//   bkg_STau_Hist.SetLineWidth(2); 
//   bkg_STau_Hist.Draw("same"); 
  
  HxcTau.Write();
  HxcTauSB.Write();
  HxScTau.Write();
  HxScTauSB.Write();
  pdf_cTauSTau_Hist2D.Write();
  sig_cTauSTau_Hist2D.Write();
  bkg_cTauSTau_Hist2D.Write();
//   pdf_cTau_Hist2D.Write();
//   sig_cTau_Hist2D.Write();
//   bkg_cTau_Hist2D.Write();
//   pdf_cTau_Hist.Write();
//   sig_cTau_Hist.Write();
//   bkg_cTau_Hist.Write();
//   pdf_STau_Hist.Write();
//   sig_STau_Hist.Write();
//   bkg_STau_Hist.Write();
  pdf_cTauSTau_X->Write();
  pdf_cTauSTau_Y->Write();
  sig_cTauSTau_X->Write();
  sig_cTauSTau_Y->Write();
  bkg_cTauSTau_X->Write();
  bkg_cTauSTau_Y->Write();
  c1->Write();
  c2->Write();
  c3->Write();
  char PDFNameMass[50] = "Bp-Mass-2016.pdf";
  char PDFNamecTau[50] = "Bp-cTau-2016.pdf";
  char PDFNameReso[50] = "Bp-Reso-2016.pdf";
  char testo[130] ;
  sprintf(testo,"mv %s %s.tmp",PDFNameMass,PDFNameMass);
  gSystem->Exec(testo);
  sprintf(testo,"mv %s %s.tmp",PDFNamecTau,PDFNamecTau);
  gSystem->Exec(testo);
  sprintf(testo,"mv %s %s.tmp",PDFNameReso,PDFNameReso);
  gSystem->Exec(testo);
  
  std::cout<<"Tau  [ps]= "<<1/(c_const*cTau->value)<<"+/-"<<cTau->error/((c_const*cTau->value)*(cTau->value))<<std::endl;
  std::cout<<"1/TauSB1 = "<<1/(tauSB1->value)<<"+/-"<<tauSB1->error/((tauSB1->value)*(tauSB1->value))<<std::endl;
  std::cout<<"1/TauSB2 = "<<1/(tauSB2->value)<<"+/-"<<tauSB2->error/((tauSB2->value)*(tauSB2->value))<<std::endl;
  std::cout<<"1/TauSB3 = "<<1/(tauSB3->value)<<"+/-"<<tauSB3->error/((tauSB3->value)*(tauSB3->value))<<std::endl;

  c1->Print(PDFNameMass);
  c2->Print(PDFNamecTau);
  c3->Print(PDFNameReso);
  OutFile->Close();
  std::cout<<"==========================================" <<std::endl;
  std::cout<<"==========================================" <<std::endl;

  stopCPU = times(&stopProc);
  gettimeofday(&stopTime, NULL);
  // Print total minimization time
  double myCPU = stopCPU - startCPU;
  double totalCPU = myCPU; 

  timersub(&stopTime, &startTime, &totalTime);
  std::cout << "Wallclock time  : " << totalTime.tv_sec + totalTime.tv_usec/1000000.0 << " seconds." << std::endl;
  std::cout << "CPU time: " << (myCPU / CLOCKS_PER_SEC) << std::endl; 
  std::cout << "Total CPU time: " << (totalCPU / CLOCKS_PER_SEC) << std::endl; 
  myCPU = stopProc.tms_utime - startProc.tms_utime;
  std::cout << "Processor time: " << (myCPU / CLOCKS_PER_SEC) << std::endl;
  std::cout<<"==========================================" <<std::endl;
  std::cout<<"==========================================" <<std::endl;
}
