#include "hip/hip_runtime.h"
#include "ExpGausProdBResoCorrPdf.hh"

EXEC_TARGET fptype device_ExpGausProdBResoCorr (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x         = evt[indices[2+indices[0]]]; 
  fptype sigmaTrue = evt[indices[3+indices[0]]]; 
  fptype mean      = p[indices[1]];
  fptype lambda    = p[indices[2]];
  fptype sigmas    = p[indices[3]];
  fptype means     = p[indices[4]];
  fptype lambdas   = p[indices[5]];
  fptype lo        = p[indices[6]];
  fptype hi        = p[indices[7]];
  fptype los       = p[indices[8]];
  fptype his       = p[indices[9]];
  fptype corrP1    = p[indices[10]];
  fptype corrP0    = p[indices[11]];
  
//  fptype sigma = p[indices[2]];
//  fptype alpha = p[indices[3]];
//  sigmaM = sigma;
    fptype sigma = (corrP1*x+corrP0)*sigmaTrue;
    if(sigma<0){
     printf("Error: ExpGausProdBResoCorrPdf sigma<0!!! ==> corrP1=%f  corrP0=%f \n", corrP1,corrP0);
     return 0;
    }
    fptype ret = 0.5*lambda; 
    fptype exparg = ret * (2*mean + lambda*sigma*sigma - 2*x);
    fptype erfarg = (mean + lambda*sigma*sigma - x) / (sigma * 1.4142135623);
    ret *= EXP(exparg); 
    ret *= ERFC(erfarg); 

    fptype y = sigma;
    fptype rets = 0.5*lambdas; 
    fptype expargs = rets * (2*means + lambdas*sigmas*sigmas - 2*y);
    fptype erfargs = (means + lambdas*sigmas*sigmas - y) / (sigmas * 1.4142135623);
//fptype exparg = lambda * (lambda*sigma*sigma/2. + mean-x);
//fptype erfarg = ((mean-x)/(sigma*sigma) + lambda)*sigma /  1.4142135623;
//
    rets *= EXP(expargs); 
    rets *= ERFC(erfargs); 
    ret=rets*ret;
  if (ret<=0){
    printf("Error: ExpGausProdBResoCorrPdf<=0!!! ==> x=%f  sigma=%f mean=%f lambda=%f sigmas=%f means=%f lambdas=%fret=%f\n", x, sigma, mean,lambda , sigmas, means,lambdas ,ret);
    printf("Error: ExpGausProdBResoCorrPdf<=0!!! ==> lo=%f  hi=%f los=%f his=%f \n", lo,hi,los,his);
    printf("Error: ExpGausProdBResoCorrPdf<=0!!! ==> corrP1=%f  corrP0=%f \n", corrP1,corrP0);
     return 0;
  }   
//
//    fptype lo = 0.01;
//    fptype hi = 0.1 ;
//fptype lo = x;
//fptype hi = x+0.00000000001 ;
//
  fptype intg_hi = 0.5*(1-EXP(lambda*(lambda*sigma*sigma/2.+mean - hi))*
                   erfc((lambda+(mean-hi)/(sigma*sigma))*sigma/1.4142135623)+ erf((hi-mean)/(sigma*1.4142135623)));   
  fptype intg_lo = 0.5*(1-EXP(lambda*(lambda*sigma*sigma/2.+mean - lo))*
                   erfc((lambda+(mean-lo)/(sigma*sigma))*sigma/1.4142135623)+ erf((lo-mean)/(sigma*1.4142135623)));  

//  fptype los = 0.0003;
//  fptype his = 0.007 ;
  fptype intg_his = 0.5*(1-EXP(lambdas*(lambdas*sigmas*sigmas/2.+means - his))*
                   erfc((lambdas+(means-his)/(sigmas*sigmas))*sigmas/1.4142135623)+ erf((his-means)/(sigmas*1.4142135623)));   
  fptype intg_los = 0.5*(1-EXP(lambdas*(lambdas*sigmas*sigmas/2.+means - los))*
                   erfc((lambdas+(means-los)/(sigmas*sigmas))*sigmas/1.4142135623)+ erf((los-means)/(sigmas*1.4142135623)));  
		    
//  
//   fptype u_hi = lambda * (hi - mean);
//   fptype v_hi = lambda * sigma;
//   fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
//   fptype intg_hi1 = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));
// //
//   fptype u_lo = lambda * (lo - mean);
//   fptype v_lo = lambda * sigma;
//   fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
//   fptype intg_lo1 = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
//
 
 
   fptype integral = fabs(intg_hi-intg_lo)*fabs(intg_his-intg_los);
//  fptype integral1 = fabs(intg_hi1-intg_lo1);
   ret = ret/integral;
 
//if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("\n\nExpGausProdBResoCorrPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral =%f\n", x, sigma, mean,lambda , ret, integral);
//  printf("ExpGausProdBResoCorrPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral1=%f\n", x, sigma, mean,lambda , ret, integral1);
//}  
 
/*  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
  printf("ExpGausProdBResoCorrPdf->integrate  sigma=%f mean=%f lambda=%f Integral=%f\n",    sigma, mean,lambda , integral);
  printf("ExpGausProdBResoCorrPdf->integrate  sigma=%f mean=%f lambda=%f IntegralLO=%f\n",  sigma, mean,lambda , intg_lo);
  printf("ExpGausProdBResoCorrPdf->integrate  sigma=%f mean=%f lambda=%f IntegralHI=%f\n",  sigma, mean,lambda , intg_hi);
  printf("ExpGausProdBResoCorrPdf->integrate  sigma=%f mean=%f lambda=%f EXP(expa_hi)=%f\n",  sigma, mean,lambda , EXP(expa_hi));
  printf("ExpGausProdBResoCorrPdf->integrate  sigma=%f mean=%f lambda=%f EXP(expa_lo)=%f\n",  sigma, mean,lambda , EXP(expa_lo));
 }
 */// printf("ExpGausProdBResoCorrPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausProdBResoCorrPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//   printf("ExpGausProdBResoCorrPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral=%f\n", x, sigma, mean,lambda , ret, integral);
//    printf("ExpGausProdBResoCorrPdf ind0 =%d  evt0 =%f \n",indices[0] ,evt[indices[0]] );
//    printf("ExpGausProdBResoCorrPdf ind1 =%d  evt1 =%f \n",indices[1] ,evt[indices[1]] );
//    printf("ExpGausProdBResoCorrPdf ind2 =%d  evt2 =%f \n",indices[2] ,evt[indices[2]] );
//    printf("ExpGausProdBResoCorrPdf ind3 =%d  evt3 =%f \n",indices[3] ,evt[indices[3]] );
//    printf("ExpGausProdBResoCorrPdf ind4 =%d  evt4 =%f \n",indices[4] ,evt[indices[4]] );
//    printf("ExpGausProdBResoCorrPdf ind5 =%d  evt5 =%f \n",indices[5] ,evt[indices[5]] );
//    printf("ExpGausProdBResoCorrPdf ind6 =%d  evt6 =%f \n",indices[6] ,evt[indices[6]] );
//    printf("ExpGausProdBResoCorrPdf ind7 =%d  evt7 =%f \n",indices[7] ,evt[indices[7]] );
//    printf("ExpGausProdBResoCorrPdf ind8 =%d  evt8 =%f \n",indices[8] ,evt[indices[8]] );
//    printf("ExpGausProdBResoCorrPdf ind9 =%d  evt9 =%f \n",indices[9] ,evt[indices[9]] );
//    printf("ExpGausProdBResoCorrPdf ind10=%d  evt10=%f \n",indices[10],evt[indices[10]] );
//    printf("ExpGausProdBResoCorrPdf ind11=%d  evt11=%f \n",indices[11],evt[indices[11]] );
//    printf("ExpGausProdBResoCorrPdf ind12=%d  evt12=%f \n",indices[12],evt[indices[12]] );
//    printf("ExpGausProdBResoCorrPdf ind13=%d  evt13=%f \n",indices[13],evt[indices[13]] );
//    printf("ExpGausProdBResoCorrPdf ind14=%d  evt14=%f \n",indices[14],evt[indices[14]] );
// }
//  return 0; 
//
  return ret;
}

MEM_DEVICE device_function_ptr ptr_to_ExpGausProdBResoCorr = device_ExpGausProdBResoCorr; 

 __host__ ExpGausProdBResoCorrPdf::ExpGausProdBResoCorrPdf (std::string n, Variable* _x, Variable* _s,  Variable* mean, Variable* lambda, Variable* sigmas, Variable* means, Variable* lambdas, 
                                             Variable* lo, Variable* hi, Variable* los, Variable* his, Variable* corrP1, Variable* corrP0) 
  : GooPdf(_x, n)
{
//   registerObservable(_x); //already registered!!!
   registerObservable(_s);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(lambda));
  pindices.push_back(registerParameter(sigmas));
  pindices.push_back(registerParameter(means));
  pindices.push_back(registerParameter(lambdas));
  pindices.push_back(registerParameter(lo));
  pindices.push_back(registerParameter(hi));
  pindices.push_back(registerParameter(los));
  pindices.push_back(registerParameter(his));
  pindices.push_back(registerParameter(corrP1));
  pindices.push_back(registerParameter(corrP0));
  GET_FUNCTION_ADDR(ptr_to_ExpGausProdBResoCorr);
  initialise(pindices); 
}

 __host__ fptype ExpGausProdBResoCorrPdf::integrate (fptype lo, fptype hi) const {
// printf("integratexxx\n");
/*  unsigned int* indices = host_indices+parameters; 
 fptype sigmaM = 0.0017; 
 fptype mean = host_params[indices[1]]  ;
 fptype lambda  = host_params[indices[2]]  ;
 fptype u_hi = lambda * (hi - mean);
 fptype v_hi = lambda * sigmaM;
// fptype v_hi = lambda * u_hi;
 fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
 fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));

 fptype u_lo = lambda * (lo - mean);
 fptype v_lo = lambda * sigmaM;
// fptype v_lo = lambda * u_lo;
 fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
 fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
 
// printf("ExpGausProdBResoCorrPdf->integrate  sigma=%f mean=%f lambda=%f Integral=%f\n",  sigmaM, mean,lambda , (intg_hi-intg_lo));
// printf("ExpGausProdBResoCorrPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausProdBResoCorrPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
 return (intg_hi-intg_lo);
 */
  return 1.;
 }
