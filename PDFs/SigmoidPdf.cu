#include "hip/hip_runtime.h"
#include "SigmoidPdf.hh"

EXEC_TARGET fptype device_Sigmoid (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x     = evt[indices[2 + indices[0]]]; 
  fptype p0 = p[indices[1]];
  fptype p1 = p[indices[2]];
  fptype p2 = p[indices[3]];
  fptype p3 = p[indices[4]];

  fptype ret = p0/(1+p3+ p1*exp(-p2*x));

//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("Sigmoid x=%f  sigma=%f mean=%f tau=%f ret=%f\n", x, sigma, mean,alpha , ret);
//  } 

  if (ret<=0) {
       printf("Sigmoid <=0!!!: x = %f , p0 = %f p1 = %f f\n",x,p0,p1,p2,p3);
       return 0.;
  }
  return ret; 
}

MEM_DEVICE device_function_ptr ptr_to_Sigmoid = device_Sigmoid; 

__host__ SigmoidPdf::SigmoidPdf (std::string n, Variable* _x, Variable* p0, Variable* p1, Variable* p2, Variable* p3) 
  : GooPdf(_x, n)
{
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(p0));
  pindices.push_back(registerParameter(p1));
  pindices.push_back(registerParameter(p2));
  pindices.push_back(registerParameter(p3));
  GET_FUNCTION_ADDR(ptr_to_Sigmoid);
  initialise(pindices); 
}


