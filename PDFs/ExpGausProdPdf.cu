#include "hip/hip_runtime.h"
#include "ExpGausProdPdf.hh"

EXEC_TARGET fptype device_ExpGausProd (fptype* evt, fptype* p, unsigned int* indices) {
  fptype x         = evt[indices[2+indices[0]]]; 
  fptype sigma     = evt[indices[3+indices[0]]]; 
  fptype mean      = p[indices[1]];
  fptype lambda    = p[indices[2]];
  fptype sigmas    = p[indices[3]];
  fptype means     = p[indices[4]];
  fptype lambdas   = p[indices[5]];
  
//  fptype sigma = p[indices[2]];
//  fptype alpha = p[indices[3]];
//  sigmaM = sigma;
    fptype ret = 0.5*lambda; 
    fptype exparg = ret * (2*mean + lambda*sigma*sigma - 2*x);
    fptype erfarg = (mean + lambda*sigma*sigma - x) / (sigma * 1.4142135623);
    ret *= EXP(exparg); 
    ret *= ERFC(erfarg); 

    fptype y = sigma;
    fptype rets = 0.5*lambdas; 
    fptype expargs = rets * (2*means + lambdas*sigmas*sigmas - 2*y);
    fptype erfargs = (means + lambdas*sigmas*sigmas - y) / (sigmas * 1.4142135623);
//fptype exparg = lambda * (lambda*sigma*sigma/2. + mean-x);
//fptype erfarg = ((mean-x)/(sigma*sigma) + lambda)*sigma /  1.4142135623;
//
    rets *= EXP(expargs); 
    rets *= ERFC(erfargs); 
    ret=rets*ret;
  if (ret<=0){
    printf("Error: ExpGausProdPdf<=0!!! ==> x=%f  sigma=%f mean=%f lambda=%f ret=%f\n", x, sigma, mean,lambda , ret);
  }   
//
    fptype lo = 0.01;
    fptype hi = 0.1 ;
//fptype lo = x;
//fptype hi = x+0.00000000001 ;
//
  fptype intg_hi = 0.5*(1-EXP(lambda*(lambda*sigma*sigma/2.+mean - hi))*
                   erfc((lambda+(mean-hi)/(sigma*sigma))*sigma/1.4142135623)+ erf((hi-mean)/(sigma*1.4142135623)));   
  fptype intg_lo = 0.5*(1-EXP(lambda*(lambda*sigma*sigma/2.+mean - lo))*
                   erfc((lambda+(mean-lo)/(sigma*sigma))*sigma/1.4142135623)+ erf((lo-mean)/(sigma*1.4142135623)));  

  fptype los = 0.0003;
  fptype his = 0.007 ;
  fptype intg_his = 0.5*(1-EXP(lambdas*(lambdas*sigmas*sigmas/2.+means - his))*
                   erfc((lambdas+(means-his)/(sigmas*sigmas))*sigmas/1.4142135623)+ erf((his-means)/(sigmas*1.4142135623)));   
  fptype intg_los = 0.5*(1-EXP(lambdas*(lambdas*sigmas*sigmas/2.+means - los))*
                   erfc((lambdas+(means-los)/(sigmas*sigmas))*sigmas/1.4142135623)+ erf((los-means)/(sigmas*1.4142135623)));  
		    
//  
//   fptype u_hi = lambda * (hi - mean);
//   fptype v_hi = lambda * sigma;
//   fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
//   fptype intg_hi1 = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));
// //
//   fptype u_lo = lambda * (lo - mean);
//   fptype v_lo = lambda * sigma;
//   fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
//   fptype intg_lo1 = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
//
 
 
   fptype integral = fabs(intg_hi-intg_lo)*fabs(intg_his-intg_los);
//  fptype integral1 = fabs(intg_hi1-intg_lo1);
   ret = ret/integral;
 
//if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//  printf("\n\nExpGausProdPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral =%f\n", x, sigma, mean,lambda , ret, integral);
//  printf("ExpGausProdPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral1=%f\n", x, sigma, mean,lambda , ret, integral1);
//}  
 
/*  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
  printf("ExpGausProdPdf->integrate  sigma=%f mean=%f lambda=%f Integral=%f\n",    sigma, mean,lambda , integral);
  printf("ExpGausProdPdf->integrate  sigma=%f mean=%f lambda=%f IntegralLO=%f\n",  sigma, mean,lambda , intg_lo);
  printf("ExpGausProdPdf->integrate  sigma=%f mean=%f lambda=%f IntegralHI=%f\n",  sigma, mean,lambda , intg_hi);
  printf("ExpGausProdPdf->integrate  sigma=%f mean=%f lambda=%f EXP(expa_hi)=%f\n",  sigma, mean,lambda , EXP(expa_hi));
  printf("ExpGausProdPdf->integrate  sigma=%f mean=%f lambda=%f EXP(expa_lo)=%f\n",  sigma, mean,lambda , EXP(expa_lo));
 }
 */// printf("ExpGausProdPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausProdPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
//  if ((0 == THREADIDX) && (0 == BLOCKIDX)){
//   printf("ExpGausProdPdf x=%f  sigma=%f mean=%f lambda=%f ret=%f integral=%f\n", x, sigma, mean,lambda , ret, integral);
//    printf("ExpGausProdPdf ind0 =%d  evt0 =%f \n",indices[0] ,evt[indices[0]] );
//    printf("ExpGausProdPdf ind1 =%d  evt1 =%f \n",indices[1] ,evt[indices[1]] );
//    printf("ExpGausProdPdf ind2 =%d  evt2 =%f \n",indices[2] ,evt[indices[2]] );
//    printf("ExpGausProdPdf ind3 =%d  evt3 =%f \n",indices[3] ,evt[indices[3]] );
//    printf("ExpGausProdPdf ind4 =%d  evt4 =%f \n",indices[4] ,evt[indices[4]] );
//    printf("ExpGausProdPdf ind5 =%d  evt5 =%f \n",indices[5] ,evt[indices[5]] );
//    printf("ExpGausProdPdf ind6 =%d  evt6 =%f \n",indices[6] ,evt[indices[6]] );
//    printf("ExpGausProdPdf ind7 =%d  evt7 =%f \n",indices[7] ,evt[indices[7]] );
//    printf("ExpGausProdPdf ind8 =%d  evt8 =%f \n",indices[8] ,evt[indices[8]] );
//    printf("ExpGausProdPdf ind9 =%d  evt9 =%f \n",indices[9] ,evt[indices[9]] );
//    printf("ExpGausProdPdf ind10=%d  evt10=%f \n",indices[10],evt[indices[10]] );
//    printf("ExpGausProdPdf ind11=%d  evt11=%f \n",indices[11],evt[indices[11]] );
//    printf("ExpGausProdPdf ind12=%d  evt12=%f \n",indices[12],evt[indices[12]] );
//    printf("ExpGausProdPdf ind13=%d  evt13=%f \n",indices[13],evt[indices[13]] );
//    printf("ExpGausProdPdf ind14=%d  evt14=%f \n",indices[14],evt[indices[14]] );
// }
//  return 0; 
//
  return ret;
}

MEM_DEVICE device_function_ptr ptr_to_ExpGausProd = device_ExpGausProd; 

 __host__ ExpGausProdPdf::ExpGausProdPdf (std::string n, Variable* _x, Variable* _s,  Variable* mean, Variable* lambda, Variable* sigmas, Variable* means, Variable* lambdas) 
  : GooPdf(_x, n)
{
//   registerObservable(_x); //already registered!!!
   registerObservable(_s);
  std::vector<unsigned int> pindices;
  pindices.push_back(registerParameter(mean));
  pindices.push_back(registerParameter(lambda));
  pindices.push_back(registerParameter(sigmas));
  pindices.push_back(registerParameter(means));
  pindices.push_back(registerParameter(lambdas));
  GET_FUNCTION_ADDR(ptr_to_ExpGausProd);
  initialise(pindices); 
}

 __host__ fptype ExpGausProdPdf::integrate (fptype lo, fptype hi) const {
// printf("integratexxx\n");
/*  unsigned int* indices = host_indices+parameters; 
 fptype sigmaM = 0.0017; 
 fptype mean = host_params[indices[1]]  ;
 fptype lambda  = host_params[indices[2]]  ;
 fptype u_hi = lambda * (hi - mean);
 fptype v_hi = lambda * sigmaM;
// fptype v_hi = lambda * u_hi;
 fptype expa_hi = -u_hi+ v_hi*v_hi*0.5+LOG(0.5*( 1+ERF((u_hi-v_hi*v_hi)/(v_hi*1.4142135623))) );
 fptype intg_hi = (0.5*(1+ERF(u_hi/(v_hi*1.4142135623))) - EXP(expa_hi));

 fptype u_lo = lambda * (lo - mean);
 fptype v_lo = lambda * sigmaM;
// fptype v_lo = lambda * u_lo;
 fptype expa_lo = -u_lo+ v_lo*v_lo*0.5+LOG(0.5*( 1+ERF((u_lo-v_lo*v_lo)/(v_lo*1.4142135623))) );
 fptype intg_lo = (0.5*(1+ERF(u_lo/(v_lo*1.4142135623))) - EXP(expa_lo));
 
// printf("ExpGausProdPdf->integrate  sigma=%f mean=%f lambda=%f Integral=%f\n",  sigmaM, mean,lambda , (intg_hi-intg_lo));
// printf("ExpGausProdPdf->host_indices  host_indices0=%d host_indices1=%d host_indices2=%d\n",  host_indices[0], host_indices[1],host_indices[2]);
// printf("ExpGausProdPdf->     indices  indices0=%d indices1=%d indices2=%d\n",  indices[0], indices[1],indices[2]);
 return (intg_hi-intg_lo);
 */
  return 1.;
 }
